#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void encryptKernel(int n, int key, char* deviceDataIn, char* deviceDataOut) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<=n) deviceDataOut[index] = deviceDataIn[index] + key;
}

__global__ void decryptKernel(int n, int key,char* deviceDataIn, char* deviceDataOut) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<=n) deviceDataOut[index] = deviceDataIn[index] - key;
}

int fileSize() {
  int size; 

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1; 
  }
  return size; 
}


int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

int writeData(int size, char *fileName, char *data) {
  ofstream file (fileName, ios::out|ios::binary|ios::trunc);
  if (file.is_open())
  {
    file.write (data, size);
    file.close();

    cout << "The entire file content was written to file." << endl;
    return 0;
  }
  else cout << "Unable to open file";

  return -1; 
}

int EncryptSeq (int n, int key, char* data_in, char* data_out) 
{  
  int i;
  timer sequentialTime = timer("Sequential encryption");
  
  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]=data_in[i] + key; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;
  
  return 0; 
}

int DecryptSeq (int n, int key, char* data_in, char* data_out)
{
  int i;
  timer sequentialTime = timer("Sequential decryption");

  sequentialTime.start();
  for (i=0; i<n; i++) { data_out[i]= data_in[i] - key; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}


int EncryptCuda (int n, int key, char* data_in, char* data_out) {
    int threadBlockSize = 1024;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char *)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char *)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char *), hipMemcpyHostToDevice));
    memoryTime.stop();
    // execute kernel
    kernelTime1.start();
    encryptKernel<<<(n/threadBlockSize)+1, threadBlockSize>>>(n, key, deviceDataIn, deviceDataOut);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char *), hipMemcpyDeviceToHost));
    memoryTime.stop();


    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int DecryptCuda (int n,  int key, char* data_in, char* data_out) {
    int threadBlockSize = 1024;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char *)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n* sizeof(char *)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char *), hipMemcpyHostToDevice));
    memoryTime.stop();
    
    // execute kernel
    kernelTime1.start();
    decryptKernel<<<(n/threadBlockSize)+1, threadBlockSize>>>(n, key, deviceDataIn, deviceDataOut);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char *), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int main(int argc, char* argv[]) {
    int n;
    int key = 1;
    n = fileSize();
    if (n == -1) {
	cout << "File not found! Exiting ... " << endl; 
	exit(0);
    }
    
    char* data_in = new char[n];
    char* data_out = new char[n];    
    char* data_in_cuda = new char[n];   
    char* data_out_cuda = new char[n];  
    readData("original.data", data_in); 

    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, key, data_in, data_out);
    writeData(n, "sequential.data", data_out);

    EncryptCuda(n, key, data_in, data_out_cuda);
    writeData(n, "cuda.data", data_out_cuda);  

    readData("cuda.data", data_in_cuda);
    readData("sequential.data",data_in);

    cout << "Decrypting a file of " << n << "characters" << endl;
    DecryptSeq(n, key, data_in, data_out);
    writeData(n, "sequential_decrypted.data", data_out);
    DecryptCuda(n, key, data_in_cuda, data_out_cuda); 
    writeData(n, "recovered.data", data_out_cuda); 
 
    delete[] data_in;
    delete[] data_out;
    
    return 0;
}
