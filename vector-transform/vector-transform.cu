#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorTransformKernel(int height, int n  , float* A, float* B, float* Result) {
 unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
 unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
 if(i < height && j < n) Result[j] = Result[j]+A[j]*B[j];

}

void vectorTransformCuda(int n, float* a, float* b, float* result) {
    int threadBlockSize = 512;
    int height = 5;
    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n*height*sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB,n*height*sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n*height*sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    timer kernelTime1 = timer("kernelTime1");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceA, a, n*height*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*height*sizeof(float), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    dim3 grid(1,n/threadBlockSize);
    dim3 block(height,threadBlockSize);

    vectorTransformKernel<<<grid, block>>>(height,n,deviceA, deviceB, deviceResult);

    hipDeviceSynchronize();
    checkCudaCall(hipGetLastError());
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(result, deviceResult, n*height*sizeof(float), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    cout << "vector-transform (kernel): \t\t" << kernelTime1  << endl;
    cout << "vector-transform (memory): \t\t" << memoryTime << endl;
}

int vectorTransformSeq(int n, float* a, float* b, float* result) {
  int i,j; 

  timer sequentialTime = timer("Sequential");
  
  sequentialTime.start();
  for (j=0; j<5; j++) {
    for (i=0; i<n; i++) {
	result[i] = result[i]+a[i]*b[i];
    }
  }
  sequentialTime.stop();
  
  cout << "vector-transform (sequential): \t\t" << sequentialTime << endl;

}

int main(int argc, char* argv[]) {
    int n = 655360;
    float* a = new float[n];
    float* b = new float[n];
    float* result = new float[n];
    float* result_s = new float[n];

    if (argc > 1) n = atoi(argv[1]);

    cout << "Iteratively transform vector A with vector B of " << n << " integer elements." << endl;
    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = 0.1*i;
	result[i]=0;
	result_s[i]=0;
    }

    vectorTransformSeq(n, a, b, result_s);
    vectorTransformCuda(n, a, b, result);
    
    // verify the resuls
    for(int i=0; i<n; i++) {
	  if (result[i]!=result_s[i]) {
      if (fabs(result[i] - result_s[i]) >0.001)
        cout << "error in results! Element " << i << " is " << result[i] << ", but should be " << result_s[i] << endl; 
            exit(1);
        }
    }
    cout << "results OK!" << endl;
            
    delete[] a;
    delete[] b;
    delete[] result;
    
    return 0;
}
