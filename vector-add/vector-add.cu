#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "cuda error: " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}

__global__ void vectorAddKernel(int n,float* A, float* B, float* Result) {
    // insert operation here
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i<n)  Result[i]= A[i] + B[i];
}

__global__ void vectorSubKernel(int n,float* A, float* B, float* Result) {
    // insert operation here
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i<n)  Result[i]= A[i] - B[i];
}

__global__ void vectorMulKernel(int n,float* A, float* B, float* Result) {
    // insert operation here
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i<n)  Result[i]= A[i] * B[i];
}

__global__ void vectorDivKernel(int n,float* A, float* B, float* Result) {
    // insert operation here
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i<n)  Result[i]= A[i] / B[i];
}



void vectorOpsCuda(int n,int operation,int gridGeoThreadBlock, float* a, float* b, float* result) {
    int threadBlockSize = 512;

    if (gridGeoThreadBlock!=0) threadBlockSize = gridGeoThreadBlock;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        std::cout << "could not allocate memory!" << std::endl;
        return;
    }

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    // execute kernel
    if(operation == 0) vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(n, deviceA, deviceB, deviceResult);
    if(operation == 1) vectorSubKernel<<<n/threadBlockSize, threadBlockSize>>>(n, deviceA, deviceB, deviceResult);
    if(operation == 2) vectorMulKernel<<<n/threadBlockSize, threadBlockSize>>>(n, deviceA, deviceB, deviceResult);
    if(operation == 3) vectorDivKernel<<<n/threadBlockSize, threadBlockSize>>>(n, deviceA, deviceB, deviceResult);
    hipDeviceSynchronize();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    high_resolution_clock::time_point t3 = high_resolution_clock::now();

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    high_resolution_clock::time_point t4 = high_resolution_clock::now();

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    if(operation == 0) std::cout << "vector-add (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    if(operation == 0) std::cout << "vector-add (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    if(operation == 0) std::cout << "vector-add (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;

    if(operation == 1) std::cout << "vector-sub (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    if(operation == 1) std::cout << "vector-sub (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    if(operation == 1) std::cout << "vector-sub (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;

    if(operation == 2) std::cout << "vector-mul (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    if(operation == 2) std::cout << "vector-mul (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    if(operation == 2) std::cout << "vector-mul (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;

    if(operation == 3) std::cout << "vector-div (H2D):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
    if(operation == 3) std::cout << "vector-div (kernel): \t\t" << duration_cast<microseconds>(t3 - t2).count() << "us" << std::endl;
    if(operation == 3) std::cout << "vector-div (D2H):    \t\t" << duration_cast<microseconds>(t4 - t3).count() << "us" << std::endl;
}

void vectorAddSeq(int n, float* a, float* b, float* result) {
    int i;

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    for (i=0; i<n; i++) {
        result[i] = a[i]+b[i];
    }

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    std::cout << "vector-add (seq):    \t\t" << duration_cast<microseconds>(t2 - t1).count() << "us" << std::endl;
}

int main(int argc, char* argv[]) {
    int n = 655360;
    float* a = new float[n];
    float* b = new float[n];
    float* result = new float[n];
    float* result_s = new float[n];
    int gridGeoThreadBlock = 0;
    int operation = 0;
    if (argc > 1) n = atoi(argv[1]);
    if (argc > 2) gridGeoThreadBlock = atoi(argv[2])
    if (argc > 3) operation = atoi(argv[3])

    std::cout << "Adding two vectors of " << n << " integer elements." << std::endl;

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = i;
    }

    vectorAddSeq(n,a, b, result_s);

    vectorOpsCuda(n, operation, gridGeoThreadBlock, a, b, result);

    // verify the resuls
    for(int i=0; i<n; i++) {
        if (result[i]!=result_s[i]) {
            std::cout << "error in results! Element " << i << " is " << result[i] << ", but should be " << result_s[i] << std::endl;
            exit(1);
        }
    }

    std::cout << "results OK!" << std::endl;

    delete[] a;
    delete[] b;
    delete[] result;

    return 0;
}
